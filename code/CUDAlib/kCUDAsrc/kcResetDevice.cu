
#include <math.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>



#include "mex.h"

#include "kcDefs.h"
#include "kcArrayFunctions.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    hipError_t ce;
    /*ce = hipSetDevice(0);
    if(ce != hipSuccess) {
        mexPrintf("Error selecting device: %d\n", (int)ce);
    }
    else {*/
    if(nrhs > 0) {
        int newDevice = (int)mxGetScalar(prhs[0]);
        kcSwitchToDevice(newDevice);
    }
    
    hipGetLastError();
        ce = hipDeviceReset();
        if(ce != hipSuccess) {
            mexPrintf("Error reseting device: %d\n", (int)ce);
        }
        else {
            int printOutput = 0;
            if(nrhs > 1) {
                printOutput = (int)mxGetScalar(prhs[1]);
            }
            if(printOutput > 0) {
                mexPrintf("Device reset.\n");
            }
        }
    //}
}
