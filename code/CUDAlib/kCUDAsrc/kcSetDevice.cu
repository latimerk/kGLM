#include "mex.h"
#include <hip/hip_runtime.h>
#include "kcDefs.h"
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    int currentDevice, newDevice;
    hipError_t ce;
    hipGetDevice(&currentDevice);
    
    printf("Current GPU device: %d\n",currentDevice);
    
    if(nrhs == 0) {
        ce = hipSetDevice(KC_GPU_DEVICE);
    }
    else {
        ce = hipSetDevice((int)mxGetScalar(prhs[0]));
    }
    if(ce != hipSuccess) {
        mexPrintf("Error selecting device ");
        mexPrintf(hipGetErrorString(ce));
        mexPrintf(" (%d)\n", (int)ce);
        mexErrMsgTxt("CUDA Errors");
    }

    
    hipGetDevice(&newDevice);
    printf("Changed to GPU device: %d\n",newDevice);

}
