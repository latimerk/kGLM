//#include "cuda/hip/hip_runtime.h"
#include <math.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "mex.h"

#include "kcDefs.h"
#include "kcArrayFunctions.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    hipError_t cudaFreeStatus;
    
    
    kcSwitchToDevice(prhs[0]);
    
	void * d_a = kcGetArrayData(prhs[0],true);
    cudaFreeStatus = hipFree(d_a);
   
    

    if(cudaFreeStatus == hipErrorInvalidDevicePointer) {
        mexPrintf("Free memory, invalid device ptr.\n");
    }
    else if(cudaFreeStatus == hipErrorNotInitialized) {
        mexPrintf("Free memory, init error.\n");
    }
 
    else if(cudaFreeStatus != hipSuccess) {
        mexPrintf("Free memory failed.\n");
    }
    else {
        unsigned long long int * out = (unsigned long long int *) mxGetPr(mxGetField(prhs[0],0,KC_ARRAY_PTR));
        *out = (unsigned long long int)0;
    }    
}
